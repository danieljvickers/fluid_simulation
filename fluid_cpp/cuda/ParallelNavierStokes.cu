#include "hip/hip_runtime.h"
//
// Created by dan on 12/10/24.
//

#include "ParallelNavierStokes.cuh"
#include <iostream>

/*
 CUDA Kernels to run on GPU
*/

template <typename T> // TODO :: Update Later
__global__ void enforce_pressure_BC_kernel(NavierStokesCell<T>* cells, int width, int height) {
    // get our location in the grid
    const int column = blockIdx.x * blockDim.x + threadIdx.x;
    const int row = blockIdx.y * blockDim.y + threadIdx.y;

    for (int c = column; c < width; c++) {
        for (int r = row; r < height; r++) {
            int index = c * height + r;

            // return if you have a BC
            if (cells[index].p_boundary_set) {
                cells[index].p_next = cells[index].p_boundary;  // enforce the BC if it has been set
                return;
            }

            // checks if you are on the edge, else do nothing
            if (r == 0) {
                cells[index].p_next = cells[index + 1].p_next; // equal to cell above
            } else if (r == width - 1) {
                cells[index].p_next = cells[index - 1].p_next; // equal to cell below
            } else if (c == 0) {
                cells[index].p_next = cells[index + height].p_next; // equal to cell to right
            } else {
                cells[index].p_next = cells[index - height].p_next;  // equal to cell to left
            }
        }
    }
}

template <typename T>
__global__ void update_pressure_kernel(NavierStokesCell<T>* cells, int width, int height) {
    // get our location in the grid
    const int column = blockIdx.x * blockDim.x + threadIdx.x + 1;
    const int row = blockIdx.y * blockDim.y + threadIdx.y + 1;

    for (int c = column; c < width - 1; c++) {
        for (int r = row; r < height - 1; r++) {
            int index = c * height + r;
            cells[index].p = cells[index].p_next;
        }
    }
}

template <typename T> // TODO :: Update Later
__global__ void enforce_velocity_BC_kernel(NavierStokesCell<T>* cells, int width, int height) {
    // get our location in the grid
    const int column = blockIdx.x * blockDim.x + threadIdx.x;
    const int row = blockIdx.y * blockDim.y + threadIdx.y;

    // iterate over u velocity values
    for (int c = column; c < width; c++) {
        for (int r = row; r < height; r++) {
            int index = c * height + r;

            // return if you have a BC
            if (cells[index].u_boundary_set) {
                cells[index].u = cells[index].u_boundary;  // enforce the BC if it has been set
                return;
            }

            // checks if you are on the edge, else do nothing
            if (r == 0) {
                cells[index].u = cells[index + 1].u; // equal to cell above
            } else if (r == width - 1) {
                cells[index].u = cells[index - 1].u; // equal to cell below
            } else if (c == 0) {
                cells[index].u = cells[index + height].u; // equal to cell to right
            } else {
                cells[index].u = cells[index - height].u;  // equal to cell to left
            }
        }
    }

    // iterate over v velocity values
    for (int c = column; c < width; c++) {
        for (int r = row; r < height; r++) {
            int index = c * height + r;

            // return if you have a BC
            if (cells[index].v_boundary_set) {
                cells[index].v = cells[index].v_boundary;  // enforce the BC if it has been set
                return;
            }

            // checks if you are on the edge, else do nothing
            if (r == 0) {
                cells[index].v = cells[index + 1].v; // equal to cell above
            } else if (r == width - 1) {
                cells[index].v = cells[index - 1].v; // equal to cell below
            } else if (c == 0) {
                cells[index].v = cells[index + height].v; // equal to cell to right
            } else {
                cells[index].v = cells[index - height].v;  // equal to cell to left
            }
        }
    }
}

template <typename T>
__global__ void unified_timestep_kernel(NavierStokesCell<T>* cells, int width, int height, T element_length_x, T element_length_y, T kinematic_viscosity, T time_step) {
    // get our location in the grid
    const int column = blockIdx.x * blockDim.x + threadIdx.x + 1;
    const int row = blockIdx.y * blockDim.y + threadIdx.y + 1;

    for (int c = column; c < width - 1; c++) {
        for (int r = row; r < height - 1; r++) {
            int index = c * height + r;
            int up = index + 1;
            int down = index - 1;
            int left = index - height;
            int right = index + height;

            // compute the central differences
            cells[index].du_dx = (cells[right].u - cells[left].u) / 2. / element_length_x;
            cells[index].dv_dx = (cells[right].v - cells[left].v) / 2. / element_length_x;
            cells[index].du_dy = (cells[up].u - cells[down].u) / 2. / element_length_y;
            cells[index].dv_dy = (cells[up].v - cells[down].v) / 2. / element_length_y;

            // compute the laplacian
            T u_laplacian = cells[left].u + cells[right].u + cells[up].u + cells[down].u;
            cells[index].u_laplacian = (u_laplacian - 4. * cells[index].u) / element_length_x / element_length_y;
            T v_laplacian = cells[left].v + cells[right].v + cells[up].v + cells[down].v;
            cells[index].v_laplacian = (v_laplacian - 4. * cells[index].v) / element_length_x / element_length_y;

            // get the time derivitives
            cells[index].du_dt = kinematic_viscosity * cells[index].u_laplacian - cells[index].u * cells[index].du_dx - cells[index].v * cells[index].du_dy;
            cells[index].dv_dt = kinematic_viscosity * cells[index].v_laplacian - cells[index].u * cells[index].dv_dx - cells[index].v * cells[index].dv_dy;

            // step forward in time
            cells[index].u_next = cells[index].u + time_step * cells[index].du_dt;
            cells[index].v_next = cells[index].v + time_step * cells[index].dv_dt;
        }
    }
}

template <typename T>
__global__ void compute_righthand_kernel(NavierStokesCell<T>* cells, int width, int height, T element_length_x, T element_length_y, T density, T time_step) {
    // get our location in the grid
    const int column = blockIdx.x * blockDim.x + threadIdx.x + 1;
    const int row = blockIdx.y * blockDim.y + threadIdx.y + 1;

    for (int c = column; c < width - 1; c++) {
        for (int r = row; r < height - 1; r++) {
            int index = c * height + r;
            int up = index + 1;
            int down = index - 1;
            int left = index - height;
            int right = index + height;

            // compute the central differences
            cells[index].du_next_dx = (cells[right].u_next - cells[left].u_next) / 2. / element_length_x;
            cells[index].dv_next_dy = (cells[up].v_next - cells[down].v_next) / 2. / element_length_y;
            cells[index].right_hand_size = (density / time_step) * (cells[index].du_next_dx + cells[index].dv_next_dy);
        }
    }
}

template <typename T>
__global__ void poisson_step_kernel(NavierStokesCell<T>* cells, int width, int height, T element_length_x, T element_length_y) {
    // get our location in the grid
    const int column = blockIdx.x * blockDim.x + threadIdx.x + 1;
    const int row = blockIdx.y * blockDim.y + threadIdx.y + 1;

    for (int c = column; c < width - 1; c++) {
        for (int r = row; r < height - 1; r++) {
            int index = c * height + r;
            int up = index + 1;
            int down = index - 1;
            int left = index - height;
            int right = index + height;

            // compute the Poisson step
            T p_next = cells[index].right_hand_size * element_length_x * element_length_y;
            p_next -= cells[left].p + cells[right].p + cells[up].p + cells[down].p;
            cells[index].p_next = p_next * -0.25;
        }
    }
}

template <typename T>
__global__ void velocity_correction_kerenl(NavierStokesCell<T>* cells, int width, int height, T element_length_x, T element_length_y, T density, T time_step) {
    // get our location in the grid
    const int column = blockIdx.x * blockDim.x + threadIdx.x + 1;
    const int row = blockIdx.y * blockDim.y + threadIdx.y + 1;

    for (int c = column; c < width - 1; c++) {
        for (int r = row; r < height - 1; r++) {
            int index = c * height + r;
            int up = index + 1;
            int down = index - 1;
            int left = index - height;
            int right = index + height;

            // compute the central differences
            cells[index].dp_dx = (cells[right].p - cells[left].p) / 2. / element_length_x;
            cells[index].dp_dy = (cells[up].p - cells[down].p) / 2. / element_length_y;

            // compute final velocity
            cells[index].u = cells[index].u_next - (time_step / density) * cells[index].dp_dx;
            cells[index].v = cells[index].v_next - (time_step / density) * cells[index].dp_dy;
        }
    }
}

/*
 Class Methods to call kernel code
*/

template <class T>
ParallelNavierStokes<T>::ParallelNavierStokes(int box_dim_x, int box_dim_y, T domain_size_x, T domain_size_y)
    : NavierStokesSolver<T>(box_dim_x, box_dim_y, domain_size_x, domain_size_y) {
    hipMalloc((void**)&this->d_cells, sizeof(NavierStokesCell<T>) * box_dim_x * box_dim_y);
    for (int x = 0; x < this->box_dimension_x; x++) {
        hipMemcpy(&(this->d_cells[x * this->box_dimension_y]), this->cells[x], sizeof(NavierStokesCell<T>) * this->box_dimension_y, hipMemcpyHostToDevice);
    }
}

template <class T>
ParallelNavierStokes<T>::~ParallelNavierStokes() {
    hipFree(this->d_cells);
}

template <class T>
void ParallelNavierStokes<T>::migrateHostToDevice() {
    for (int x = 0; x < this->box_dimension_x; x++) {
        hipMemcpy(&(this->d_cells[x * this->box_dimension_y]), this->cells[x], sizeof(NavierStokesCell<T>) * this->box_dimension_y, hipMemcpyHostToDevice);
    }
}


template <class T>
void ParallelNavierStokes<T>::migrateDeviceToHost() {
    for (int x = 0; x < this->box_dimension_x; x++) {
        hipMemcpy(this->cells[x], &(this->d_cells[x * this->box_dimension_y]), sizeof(NavierStokesCell<T>) * this->box_dimension_y, hipMemcpyDeviceToHost);
    }
}


template <class T>
void ParallelNavierStokes<T>::solve() {
    // loop over each time step
    for (int i = 0; i < this->num_iterations; i++) {
        this->unifiedApproximateTimeStep();
        this->unifiedComputeRightHand();

        // take a series of poisson steps to approximate the pressure in each cell
        for (int j = 0; j < this->num_poisson_iterations; j++) {
            // compute the Poisson step, enforce BCs, and enforce the pressure
            this->computePoissonStepApproximation();
            this->enforcePressureBoundaryConditions();
            this->updatePressure();
        }

        // get the pressure central difference, correct the u and v values, and enforce BCs
        this->unifiedVelocityCorrection();
        this->enforceVelocityBoundaryConditions();
    }
}

template <class T>
void ParallelNavierStokes<T>::migrateSolve() {
    this->migrateHostToDevice();
    this->solve();
    this->migrateDeviceToHost();
}

template <class T>
void ParallelNavierStokes<T>::enforcePressureBoundaryConditions() {
    dim3 block_size(KERNEL_2D_WIDTH, KERNEL_2D_HEIGHT);  // compute the size of each block
    int bx = (this->box_dimension_x + block_size.x - 1) / block_size.x;  // x size in blocks of the grid
    int by = (this->box_dimension_y + block_size.y - 1) / block_size.y;  // y size in blocks of the grid
    dim3 grid_size = dim3(bx, by);

    enforce_pressure_BC_kernel<T><<<grid_size, block_size>>>(this->d_cells, this->box_dimension_x, this->box_dimension_y);
}

template <class T>
void ParallelNavierStokes<T>::updatePressure() {
    dim3 block_size(KERNEL_2D_WIDTH, KERNEL_2D_HEIGHT);  // compute the size of each block
    int bx = (this->box_dimension_x + block_size.x - 3) / block_size.x;  // x size in blocks of the grid
    int by = (this->box_dimension_y + block_size.y - 3) / block_size.y;  // y size in blocks of the grid
    dim3 grid_size = dim3(bx, by);

    update_pressure_kernel<T><<<grid_size, block_size>>>(this->d_cells, this->box_dimension_x, this->box_dimension_y);
}

template <class T>
void ParallelNavierStokes<T>::enforceVelocityBoundaryConditions() {
    dim3 block_size(KERNEL_2D_WIDTH, KERNEL_2D_HEIGHT);  // compute the size of each block
    int bx = (this->box_dimension_x + block_size.x - 1) / block_size.x;  // x size in blocks of the grid
    int by = (this->box_dimension_y + block_size.y - 1) / block_size.y;  // y size in blocks of the grid
    dim3 grid_size = dim3(bx, by);

    enforce_velocity_BC_kernel<T><<<grid_size, block_size>>>(this->d_cells, this->box_dimension_x, this->box_dimension_y);
}

template <class T>
void ParallelNavierStokes<T>::unifiedApproximateTimeStep() {
    dim3 block_size(KERNEL_2D_WIDTH, KERNEL_2D_HEIGHT);  // compute the size of each block
    int bx = (this->box_dimension_x + block_size.x - 3) / block_size.x;  // x size in blocks of the grid
    int by = (this->box_dimension_y + block_size.y - 3) / block_size.y;  // y size in blocks of the grid
    dim3 grid_size = dim3(bx, by);

    unified_timestep_kernel<T><<<grid_size, block_size>>>(this->d_cells, this->box_dimension_x, this->box_dimension_y,
        this->element_length_x,this->element_length_y, this->kinematic_viscosity, this->time_step);
}

template <class T>
void ParallelNavierStokes<T>::unifiedComputeRightHand() {
    dim3 block_size(KERNEL_2D_WIDTH, KERNEL_2D_HEIGHT);  // compute the size of each block
    int bx = (this->box_dimension_x + block_size.x - 3) / block_size.x;  // x size in blocks of the grid
    int by = (this->box_dimension_y + block_size.y - 3) / block_size.y;  // y size in blocks of the grid
    dim3 grid_size = dim3(bx, by);

    compute_righthand_kernel<T><<<grid_size, block_size>>>(this->d_cells, this->box_dimension_x, this->box_dimension_y,
        this->element_length_x, this->element_length_y, this->density, this->time_step);
}

template <class T>
void ParallelNavierStokes<T>::computePoissonStepApproximation() {
    dim3 block_size(KERNEL_2D_WIDTH, KERNEL_2D_HEIGHT);  // compute the size of each block
    int bx = (this->box_dimension_x + block_size.x - 3) / block_size.x;  // x size in blocks of the grid
    int by = (this->box_dimension_y + block_size.y - 3) / block_size.y;  // y size in blocks of the grid
    dim3 grid_size = dim3(bx, by);

    poisson_step_kernel<T><<<grid_size, block_size>>>(this->d_cells, this->box_dimension_x, this->box_dimension_y,
        this->element_length_x, this->element_length_y);
}

template <class T>
void ParallelNavierStokes<T>::unifiedVelocityCorrection() {
    dim3 block_size(KERNEL_2D_WIDTH, KERNEL_2D_HEIGHT);  // compute the size of each block
    int bx = (this->box_dimension_x + block_size.x - 3) / block_size.x;  // x size in blocks of the grid
    int by = (this->box_dimension_y + block_size.y - 3) / block_size.y;  // y size in blocks of the grid
    dim3 grid_size = dim3(bx, by);

    velocity_correction_kerenl<T><<<grid_size, block_size>>>(this->d_cells, this->box_dimension_x, this->box_dimension_y,
        this->element_length_x, this->element_length_y, this->density, this->time_step);
}
