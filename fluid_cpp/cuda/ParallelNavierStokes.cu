//
// Created by dan on 12/10/24.
//

#include "ParrallelNavierStokes.cuh"

template <class T>
ParrallelNavierStokes<T>::ParrallelNavierStokes(int box_dim_x, int box_dim_y, T domain_size_x, T domain_size_y)
    : NavierStokesSolver<T>(box_dim_x, box_dim_y, domain_size_x, domain_size_y) {
    hipMalloc((void**)this->d_cells, sizeof(NavierStokesCell<T>) * box_dim_x * box_dim_y);
    cudaMemCpy(this->d_cells, this->cells, sizeof(NavierStokesCell<T>) * box_dim_x * box_dim_y, cudaMemCpyHostToDevice);
}

template <class T>
ParallelNavierStokes<T>::~ParallelNavierStokes() {
    hipFree(this->d_cells);
}

template <class T>
ParrallelNavierStokes<T>::~SerielNavierStokes() {

}